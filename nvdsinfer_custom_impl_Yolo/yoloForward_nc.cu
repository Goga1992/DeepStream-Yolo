/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void gpuYoloLayer_nc(const float* input, float* output, int* count, const uint netWidth, const uint netHeight,
    const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes, const float scaleXY,
    const float* anchors, const int* mask)
{
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
    return;

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  const float objectness = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

  const float alpha = scaleXY;
  const float beta = -0.5 * (scaleXY - 1);

  float xc = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)] * alpha + beta + x_id) * netWidth /
      gridSizeX;

  float yc = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)] * alpha + beta + y_id) * netHeight /
      gridSizeY;

  float w = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] * 2, 2) * anchors[mask[z_id] * 2];

  float h = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] * 2, 2) * anchors[mask[z_id] * 2 + 1];

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int _count = (int)atomicAdd(count, 1);

  output[_count * 7 + 0] = xc;
  output[_count * 7 + 1] = yc;
  output[_count * 7 + 2] = w;
  output[_count * 7 + 3] = h;
  output[_count * 7 + 4] = maxProb;
  output[_count * 7 + 5] = maxIndex;
  output[_count * 7 + 6] = objectness;
}

hipError_t cudaYoloLayer_nc(const void* input, void* output, void* count, const uint& batchSize, uint64_t& inputSize,
    uint64_t& outputSize, const uint& netWidth, const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY,
    const uint& numOutputClasses, const uint& numBBoxes, const float& scaleXY, const void* anchors, const void* mask,
    hipStream_t stream);

hipError_t cudaYoloLayer_nc(const void* input, void* output, void* count, const uint& batchSize, uint64_t& inputSize,
    uint64_t& outputSize, const uint& netWidth, const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY,
    const uint& numOutputClasses, const uint& numBBoxes, const float& scaleXY, const void* anchors, const void* mask,
    hipStream_t stream)
{
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1, (gridSizeY / threads_per_block.y) + 1,
      (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuYoloLayer_nc<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float*> (input) + (batch * inputSize),
        reinterpret_cast<float*> (output) + (batch * 7 * outputSize),
        reinterpret_cast<int*> (count) + (batch),
        netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, scaleXY,
        reinterpret_cast<const float*> (anchors), reinterpret_cast<const int*> (mask));
  }
  return hipGetLastError();
}
