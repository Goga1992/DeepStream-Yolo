/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__device__ void softmaxGPU(const float* input, const int bbindex, const int numGridCells, uint z_id,
    const uint numOutputClasses, float temp, float* output)
{
  int i;
  float sum = 0;
  float largest = -INFINITY;
  for (i = 0; i < numOutputClasses; ++i) {
    int val = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    largest = (val>largest) ? val : largest;
  }
  for (i = 0; i < numOutputClasses; ++i) {
    float e = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] / temp - largest / temp);
    sum += e;
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] = e;
  }
  for (i = 0; i < numOutputClasses; ++i) {
    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
  }
}

__global__ void gpuRegionLayer(const float* input, float* softmax, float* output, int* count, const uint netWidth,
    const uint netHeight, const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes,
    const float* anchors)
{
  uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
  uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
  uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

  if (x_id >= gridSizeX || y_id >= gridSizeY || z_id >= numBBoxes)
    return;

  const int numGridCells = gridSizeX * gridSizeY;
  const int bbindex = y_id * gridSizeX + x_id;

  const float objectness = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

  float xc = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]) + x_id) * netWidth / gridSizeX;

  float yc = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]) + y_id) * netHeight / gridSizeY;

  float w = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) * anchors[z_id * 2] * netWidth /
      gridSizeX;

  float h = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) * anchors[z_id * 2 + 1] * netHeight /
      gridSizeY;

  softmaxGPU(input, bbindex, numGridCells, z_id, numOutputClasses, 1.0, softmax);

  float maxProb = 0.0f;
  int maxIndex = -1;

  for (uint i = 0; i < numOutputClasses; ++i) {
    float prob = softmax[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
    if (prob > maxProb) {
      maxProb = prob;
      maxIndex = i;
    }
  }

  int _count = (int)atomicAdd(count, 1);

  output[_count * 7 + 0] = xc;
  output[_count * 7 + 1] = yc;
  output[_count * 7 + 2] = w;
  output[_count * 7 + 3] = h;
  output[_count * 7 + 4] = maxProb;
  output[_count * 7 + 5] = maxIndex;
  output[_count * 7 + 6] = objectness;
}

hipError_t cudaRegionLayer(const void* input, void* softmax, void* output, void* count, const uint& batchSize,
    uint64_t& inputSize, uint64_t& outputSize, const uint& netWidth, const uint& netHeight, const uint& gridSizeX,
    const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes, const void* anchors, hipStream_t stream);

hipError_t cudaRegionLayer(const void* input, void* softmax, void* output, void* count, const uint& batchSize,
    uint64_t& inputSize, uint64_t& outputSize, const uint& netWidth, const uint& netHeight, const uint& gridSizeX,
    const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes, const void* anchors, hipStream_t stream)
{
  dim3 threads_per_block(16, 16, 4);
  dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1, (gridSizeY / threads_per_block.y) + 1,
      (numBBoxes / threads_per_block.z) + 1);

  for (unsigned int batch = 0; batch < batchSize; ++batch) {
    gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
        reinterpret_cast<const float*> (input) + (batch * inputSize),
        reinterpret_cast<float*> (softmax) + (batch * inputSize),
        reinterpret_cast<float*> (output) + (batch * 7 * outputSize),
        reinterpret_cast<int*> (count) + (batch),
        netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
        reinterpret_cast<const float*> (anchors));
  }
  return hipGetLastError();
}
